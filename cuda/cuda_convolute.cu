#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "cuda_convolute.h"
#include "funcs.h"
#include "hip/hip_runtime.h"
#include ""

/* CUDA kernel. Each thread takes care of one element of src */
__global__ void kernel_conv_grey(uint8_t *src, uint8_t *dst, int width, int height) {
	int i, j, k, l;
	/* Init static filter */
	int h[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
	/* get position */
	size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	/* convolute */
	if (0 < x && x < height-1 && 0 < y && y < width-1) {
		float val = 0;
		for (i = x-1, k = 0 ; i <= x+1 ; i++, k++)
			for (j = y-1, l = 0 ; j <= y+1 ; j++, l++)
				val += src[width * i + j] * h[k][l] / 16.0;
		dst[width * x + y] = val;
	}
}

__global__ void kernel_conv_rgb(uint8_t *src, uint8_t *dst, int width, int height) {
	int i, j, k, l;
	/* Init static filter */
	int h[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
	/* get position */
	size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	/* convolute */
	if (0 < x && x < height-1 && 0 < y && y < 3*width-3) {
		float redval = 0, greenval = 0, blueval = 0;
		for (i = x-1, k = 0 ; i <= x+1 ; i++, k++) {
			for (j = (y*3)-3, l = 0 ; j <= (y*3)+3 ; j+=3, l++) {
				redval += src[(width*3) * i + j]* h[k][l] /16.0;
				greenval += src[(width*3) * i + j+1] * h[k][l] /16.0;
				blueval += src[(width*3) * i + j+2] * h[k][l] /16.0;
			}
		}
		dst[width*3 * x + (y*3)] = redval;
		dst[width*3 * x + (y*3)+1] = greenval;
		dst[width*3 * x + (y*3)+2] = blueval;
	}
}

extern "C" void gpuConvolute(uint8_t *src, int width, int height, int loops, color_t imageType)
{
	/* Device vectors */
	uint8_t *d_src, *d_dst, *tmp;
	size_t bytes = (imageType == GREY) ? height * width : height * width*3;

	/* Allocate memory for each vector on GPU */
    CUDA_SAFE_CALL( hipMalloc(&d_src, bytes * sizeof(uint8_t)) );
    CUDA_SAFE_CALL( hipMalloc(&d_dst, bytes * sizeof(uint8_t)) );
 
    /* Copy host vectors to device memory */
    CUDA_SAFE_CALL( hipMemcpy(d_src, src, bytes, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemset(d_dst, 0, bytes) );

    int t;
	const int blockSize = 16;
	/* Convolute "loops" times */
	for (t = 0 ; t < loops ; t++) {
		
		if (imageType == GREY) {
			/* Specify layout of Grid and Blocks */
			int gridX = FRACTION_CEILING(height, blockSize);
			int gridY = FRACTION_CEILING(width, blockSize);
			dim3 block(blockSize, blockSize);
			dim3 grid(gridX, gridY);
			kernel_conv_grey<<<grid, block>>>(d_src, d_dst, width, height);
		} else if (imageType == RGB) {
			int gridX = FRACTION_CEILING(height, blockSize);
			int gridY = FRACTION_CEILING(width*3, blockSize);
			dim3 block(blockSize, blockSize);
			dim3 grid(gridX, gridY);
			kernel_conv_rgb<<<grid, block>>>(d_src, d_dst, width, height);
		}

		/* swap arrays */
		tmp = d_src;
	    d_src = d_dst;
	    d_dst = tmp;
	}

	CUDA_SAFE_CALL( hipGetLastError() );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    
    /* Copy array back to host */
    if (loops%2 == 0) {
    	CUDA_SAFE_CALL( hipMemcpy(src, d_src, bytes, hipMemcpyDeviceToHost) );
    } else {
   		CUDA_SAFE_CALL( hipMemcpy(src, d_dst, bytes, hipMemcpyDeviceToHost) );
   	}

	// Release device memory
    CUDA_SAFE_CALL( hipFree(d_src) );
    CUDA_SAFE_CALL( hipFree(d_dst) );
}
